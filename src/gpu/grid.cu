#include "hip/hip_runtime.h"
/*
 * grid.cpp
 *
 *  Created on: 17/03/2018
 *      Author: phr
 */

#include "grid.h"

#include <iostream>

grid::grid(int type, int size) {
  graph_size = size;
  numEdges = graph_size - 1;
  numRibs = graph_size - 2;

  // Each pixel has 2
  CVDPosition = (float *)malloc(graph_size * 2 * sizeof(float));
  hipMallocManaged(&Location, graph_size * 2 * sizeof(float));

  // do graphnodes
  if (type == PROTANOPE) {
    // PROTANOPE LIMITS
    A = {8.648425, -73.086372, 56.664734};
    C = {-14.907598, 86.293831, 89.536812};
  }

  miAB = A[1] / A[0];
  miBC = C[1] / C[0];

  float x_step = fabs((A[0] - C[0]) / (graph_size * 1.0));
  float x_start = C[0];

  for (int i = 0; i < graph_size; i++) {
    Location[pX(i)] = (x_start <= 0) ? miBC * x_start : miAB * x_start;
    Location[pY(i)] = -x_start;
    CVDPosition[pX(i)] = x_start;
    CVDPosition[pY(i)] = (x_start <= 0) ? miBC * x_start : miAB * x_start;
    x_start += x_step;
  }

  // Create Elastic Map Edges - Each edge is composed by two parts
  Edges = (int *)malloc(numEdges * 2 * sizeof(int));
  for (int i = 0; i < numEdges; i++) {
    Edges[i * 2 + 0] = i;
    Edges[i * 2 + 1] = i + 1;
  }

  // Create Elastic Map Ribs - Each ribs is composed by three parts
  Ribs = (int *)malloc(numRibs * 3 * sizeof(int));
  for (int i = 0; i < numRibs; i++) {
    Ribs[i * 3 + 0] = i + 1;
    Ribs[i * 3 + 1] = i;
    Ribs[i * 3 + 2] = i + 2;
  }
}