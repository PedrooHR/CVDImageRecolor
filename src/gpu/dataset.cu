/*
 * dataset.cpp
 *
 *  Created on: 17/03/2018
 *      Author: phr
 */

#include "dataset.h"

dataset::dataset(const char *imgPath) {
  cimg_library::CImg<unsigned int> image(imgPath);
  width = image.width();
  height = image.height();
  int size = width * height;
  
  hipMallocManaged(&Datapoints, size * 3 * sizeof(float));

  for (int i = 0; i < size; i++) {
    int x = i % width;
    int y = i / width;
    // create Lab
    float *lab_color =
        getLabColor(image(x, y, 0), image(x, y, 1), image(x, y, 2));

    // Fill data points [0: L; 1: a; 2: b]
    Datapoints[L(i)] = lab_color[0];
    Datapoints[a(i)] = lab_color[1];
    Datapoints[b(i)] = lab_color[2];
  };
  Datasize = size;
}
